#include "hip/hip_runtime.h"
// nvcc cuda.cu -o cuda
#include <bits/stdc++.h>
#include <sys/time.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

using namespace std;
typedef __int128 i128;
#define MAX_NODES 10000

i128 ceil_division(i128 num, i128 den){
    return (num + den - (i128)1) / den;
}

i128 hostFastExpo( i128 base, i128 expo, i128 m ){
    if( expo == 0 ) return 1;
    i128 prv_expo = expo/(i128)2;
    i128 ret = hostFastExpo(base, prv_expo, m);
    ret = (ret * ret)%m;
    if( expo%(i128)2 ) ret = (ret * base)%m;
    return ret;
}

__device__ i128 fastExpo( i128 base, i128 expo, i128 m ){
    i128 ret = 1, current = base;
    while(expo > 0){
        int r = expo%((i128) 2);
        if( r == 1 ) ret = (ret*current)%m;
        expo /= (i128) 2;
        current = (current * current)%m;
    }
    return ret;
}

__device__ i128 function_1( i128 a, i128 n, i128 p, i128 m ){
    i128 expo = (n * p)%m;
    return fastExpo(a, expo, m);
}

__device__ i128 function_2( i128 a, i128 b, i128 q, i128 m ){
    i128 ret = fastExpo(a, q, m);
    ret = (ret * b)%m;
    return ret;
}

__global__ void calculateFunction1(long long *a, long long *m, long long *n, long long *limit, long long *step, 
                                    long long *results, long long *keys){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = gridDim.x * blockDim.x;
    //printf("IN THREAD %d\n", threadId);

    i128 low = (*step) * threadId + (i128)1;
    i128 high = low + (*step) - (i128)1;
    if( threadId + 1 == numThreads ) high = (*limit);
    long long results_index = threadId * (*step);
    //printf("MIDDLE THREAD %d :: lo=%lld hi=%lld\n", threadId, (long long)low, (long long)high);
    for(i128 p = low ; p <= high ; p ++) {
        i128 value = function_1((*a), (*n), p, (*m));
        // if((p%(i128)10000) == 0) printf("\t --> %d TRY p = %lld\n",threadId, (long long)p);
        // INSERT TO RESULTS
        //printf("\tPut %d -> %d in %d\n", (int) p, (int) value, (int) results_index);
        results[results_index] = (long long) p;
        keys[results_index] = (long long) value;
        results_index ++;
    }
    //printf("OUT THREAD %d\n", threadId);
}

__device__ long long getEqualResult(long long limit, long long *results, long long *keys, long long target){
    int low = 0;
    int high = limit-1;
    int middle;
    while( low < high ){
        middle = (low+high+1)/2;
        if( keys[middle] <= target ) low = middle;
        else high = middle-1;
    }
    if(keys[low] == target) return results[low];
    return -1;
}

__global__ void calculateFunction2(long long *a, long long *b, long long *m, long long *n,
                                    long long *limit, long long *array_limit, long long *step, 
                                    long long *results, long long *keys, long long* x){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = gridDim.x * blockDim.x;
    // //printf("IN F2 THREAD %d\n", threadId);

    i128 low = (*step) * threadId;
    i128 high = low + (*step) - (i128)1;
    if( threadId + 1 == numThreads ) high = (*limit);
    for(i128 q = low ; q <= high ; q ++) {
        long long value = (long long)function_2((*a), (*b), q, (*m));
        long long findP = getEqualResult((*array_limit), results, keys, value);
        // //printf("\tTry %d -> %d :: %d\n", (int) q, (int) value, (int) findP);
        if( findP == -1 ) continue;
        i128 currentX = ((i128)(*n) * (i128)findP)%( (i128)(*m) );
        currentX = (currentX - q + (i128)(*m))%( (i128)(*m) );
        *x = (long long) currentX;
        // printf("\t FOUND X :: p=%d q=%d x=%d :: value=%d\n",(int)findP, (int)q, (int)currentX, (int) value);
    }
    // //printf("OUT F2 THREAD %d\n", threadId);
}

__global__ void print_arrays(long long *limit, long long *results, long long *keys){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("IN PRINT THREAD %d\n", threadId);
    for(int i = 0 ; i < (*limit) ; i ++){
        printf("\tpos %d -> %d :: %d\n",i, (int)keys[i], (int)results[i]);
    }
    printf("OUT PRINT THREAD %d\n", threadId);
}

int main(int argc, char* argv[]){

    i128 a, b, m;
    long long read_a, read_b, read_m;
    bool verbose_flag = 0;
    int blocks = 1, threadsPerBlock = 1;

    stringstream read_a_SS(argv[1]);
    read_a_SS >> read_a;

    stringstream read_b_SS(argv[2]);
    read_b_SS >> read_b;

    stringstream read_m_SS(argv[3]);
    read_m_SS >> read_m;

    stringstream read_flag_SS(argv[4]);
    read_flag_SS >> verbose_flag;
    
    stringstream blocks_SS(argv[5]);
    blocks_SS >> blocks;

    stringstream threads_SS(argv[6]);
    threads_SS >> threadsPerBlock;

    a = read_a;
    b = read_b;
    m = read_m;

    int numThreads = blocks * threadsPerBlock;

    if(verbose_flag){
        cout << "Solve " << (long long)a << "^x" << " = " << (long long)b
            << " mod " << (long long)m << endl ;
        cout << "Number of blocks: " << blocks << endl ;
        cout << "Threads per block: " << threadsPerBlock << endl ;
        cout << "Total threads: " << numThreads << endl ;
    }

    // Host variables
    long long host_a = a;
    long long host_b = b;
    long long host_m = m;
    long long host_n = sqrt((long double) m);
    threadsPerBlock = min( threadsPerBlock , (int) (host_n/(long long)blocks) );
    numThreads = blocks * threadsPerBlock;
    long long host_limit = ceil_division(host_m,host_n);
    long long host_step = host_limit / (long long) numThreads;
    long long host_x = -1;

    // cout << "host_n = " << host_n << endl ;

    int var_size = sizeof( long long );
    int array_value_size = host_n + 5;
    int array_size = array_value_size * sizeof( long long );
    // cout << "array_size = " << array_size << endl ;

    // Device variables
    long long *device_a;
    long long *device_b;
    long long *device_m;
    long long *device_n;
    long long *device_limit;
    long long *device_step;
    long long *device_results;
    long long *device_keys;
    long long *device_x;
    long long *device_array_limit;

    hipMalloc( (void**)&device_a , var_size );
    hipMalloc( (void**)&device_b , var_size );
    hipMalloc( (void**)&device_m , var_size );
    hipMalloc( (void**)&device_n , var_size );
    hipMalloc( (void**)&device_limit , var_size );
    hipMalloc( (void**)&device_step , var_size );
    hipMalloc( (void**)&device_results , array_size*2 );
    hipMalloc( (void**)&device_keys , array_size*2 );
    hipMalloc( (void**)&device_x , var_size );
    hipMalloc( (void**)&device_array_limit , var_size );

    hipMemcpy( device_a , &host_a , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_b , &host_b , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_m , &host_m , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_n , &host_n , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_limit , &host_limit , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_step , &host_step , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_x , &host_x , var_size , hipMemcpyHostToDevice );

    hipDeviceSynchronize();

    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);

    // Call the Function 1 kernel
    calculateFunction1<<<blocks,threadsPerBlock>>>(
        device_a, device_m, device_n, device_limit, device_step,
        device_results, device_keys);
    hipDeviceSynchronize();

    // cout << "AFTER F1" << endl ;

    // Check arrays after process
    // print_arrays<<<1,1>>>(device_limit, device_results, device_keys);
    // hipDeviceSynchronize();

    // cout << "AFTER C1" << endl ;

    // Sort results and keys
    thrust::device_ptr<long long> thrust_keys(device_keys);
    thrust::device_ptr<long long> thrust_results(device_results);
    thrust::sort_by_key(thrust::device, thrust_keys, thrust_keys + host_limit, thrust_results);
    hipDeviceSynchronize();

    // Check sort process
    // print_arrays<<<1,1>>>(device_limit, device_results, device_keys);
    // hipDeviceSynchronize();

    // Call the Function 2 kernel
    hipMemcpy( device_array_limit , &host_limit , var_size , hipMemcpyHostToDevice );
    host_limit = host_n;
    host_step = host_limit / (long long) numThreads;
    hipMemcpy( device_limit , &host_limit , var_size , hipMemcpyHostToDevice );
    hipMemcpy( device_step , &host_step , var_size , hipMemcpyHostToDevice );

    calculateFunction2<<<blocks,threadsPerBlock>>>(
        device_a, device_b, device_m, device_n, device_limit, device_array_limit, device_step,
        device_results, device_keys, device_x);
    hipDeviceSynchronize();

    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);

    hipMemcpy( &host_x , device_x , var_size , hipMemcpyDeviceToHost );
    // cout << "FOUND X " << host_x << " :: " << host_a  << " ^ " << host_x << " =? " << host_b << " mod " << host_m << "\n" ;
    // cout << host_a  << " ^ " << host_x << " = " << (long long)hostFastExpo(a,host_x,m) << "\n" ;

    // Free device memory
    hipFree( device_a );
    hipFree( device_b );
    hipFree( device_m );
    hipFree( device_n );
    hipFree( device_limit );
    hipFree( device_step );
    hipFree( device_results );
    hipFree( device_keys );
    hipFree( device_x );
    hipFree( device_array_limit );

    if(verbose_flag)
        cout << (long long)a << "^" << (long long)host_x << " = " << (long long)b
            << " mod " << (long long)m << endl ;

    assert(hostFastExpo(a,host_x,m) == b);
    printf("%ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

    //printf("POST TEST\n");
    
}